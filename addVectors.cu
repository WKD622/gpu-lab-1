#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <helper_timer.h>
#include <time.h>

__global__ void add(int *a, int *b, int *c, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}

void cpu_add(int *a, int *b, int *c, int N)
{
    int i;
    for (i = 0; i < N; i++)
    {
        c[i] = a[i] + b[i];
    }
}

bool check_ans(int *c, int *dev_c, int N)
{
    int i;
    bool are_right = true;

    for (i = 0; i < N; i++)
    {
        are_right = are_right && (c[i] == dev_c[i]);
    }

    return are_right;
}

int main(int argc, char *argv[])
{
    int N = atoi(argv[1]);
    int block_size = atoi(argv[2]);
    int grid_size = (int)ceil((float)N / block_size);
    int *a, *b, *c, *c_cpu, *dev_a, *dev_b, *dev_c;
    clock_t start, end;
    float cpu_time;
    a = (int *)malloc(N * sizeof(int));
    b = (int *)malloc(N * sizeof(int));
    c = (int *)malloc(N * sizeof(int));
    c_cpu = (int *)malloc(N * sizeof(int));
    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i * 2;
    }
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice);

    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    add<<<grid_size, block_size>>>(dev_a, dev_b, dev_c, N);

    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    float gpu_time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    start = clock();
    cpu_add(a, b, c_cpu, N);
    end = clock();
    cpu_time = ((float)(end - start) / CLOCKS_PER_SEC) * 1000.0;

    if (check_ans(c_cpu, c, N))
    {
        printf("gpu time: %f, cpu time: %f, blocksize %d, grid size:%d\n", gpu_time, cpu_time, block_size, grid_size);
    }
    else
    {
        printf("gpu time: %f, cpu time: %f, blocksize %d, grid size:%d\n", gpu_time, cpu_time, block_size, grid_size);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}